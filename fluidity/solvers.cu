#include "hip/hip_runtime.h"
#include "cudastatic.hpp"
#include <sys/time.h>

// Texture references for CSR matrix 
texture<int,1> tex_findrm, tex_colm;
texture<int2,1> tex_val;

// Scratchpad used by vector dot product for reduction
double* scratchpad;

// Kernel block and grid parameters - threads in a block and blocks in a grid
#define NUM_THREADS 128
#define NUM_BLOCKS 128

// Solver parameters - relative tolerance and maximum iterations
// FIXME this should really be read from the flml!
#define epsilon2 1e-14
#define IMAX 300

// For timing solver
double utime () {
  struct timeval tv;

  gettimeofday (&tv, NULL);

  return (tv.tv_sec + double (tv.tv_usec) * 1e-6);
}

__device__ int eleId(int ele, int node, int n) {
  return node*n+ele;
}

__device__ int lmatIdx(int a, int b, int c, int n) {
  return a*3*n+b*n+c;
}

// Creates a diagonal matrix stored in a vector pcmat, from the CSR matrix findrm, colm, val.
// n is the matrix size.
static __device__ double fetch_double(texture<int2,1> val, int elem)
{
  int2 v = tex1Dfetch(val, elem);
  return __hiloint2double(v.y, v.x);
}

__global__ void create_jac_sym(int n, int* findrm, int* colm, double* val, double* pcmat)
{
  int k, elem;

  for(elem=THREAD_ID; elem<n; elem+=THREAD_COUNT)
    for(k=findrm[elem]-1; k<findrm[elem+1]-1; k++)
      if(colm[k]==elem+1)
        pcmat[elem] = 1.0/val[k];
}

// Multiplies diagonal matrix mat stored as a vector by the vector src, storing result in dest.
// n is the vector length.
__global__ void diag_spmv(int n, double *mat, double *src, double *dest)
{
  int elem;

  for (elem=THREAD_ID; elem<n; elem+=THREAD_COUNT)
    dest[elem] = mat[elem]*src[elem];
}

// Sets the length-n vector vec to the zero vector.
__global__ void veczero(int n, double* vec)
{
  int elem;

  for (elem=THREAD_ID; elem<n; elem+=THREAD_COUNT)
    vec[elem] = 0;
}

// Allows fetching double values from texture memory, which only supports integers

// Multiplies the CSR matrix in texture memory tex_findrm, tex_colm, tex_val by src and stores the
// result in dest. n is the matrix size/vector length.
__global__ void csr_spmv(int n, double* src, double* dest, int *findrm)
{
  int elem;

  for (elem=THREAD_ID; elem<n; elem+=THREAD_COUNT) {
    dest[elem] = 0;
    int a=findrm[elem];
    int b=findrm[elem+1];
    for (int k=a;k<b;k++)
      dest[elem] += fetch_double(tex_val,k-1)*src[tex1Dfetch(tex_colm,k-1)-1];

  }
}

// Computes the dot product of length-n vectors vec1 and vec2. This is reduced in tmp into a
// single value per thread block. The reduced value is stored in the array partial.
__global__ void vecdot_partial(int n, double* vec1, double* vec2, double* partial)
{
  int elem;
  __shared__ double tmp[512];
  tmp[threadIdx.x] = 0;

  for (elem=THREAD_ID; elem<n; elem+=THREAD_COUNT)
    tmp[threadIdx.x] += vec1[elem]*vec2[elem];

  for (int i=blockDim.x/2;i>=1;i = i/2) {
    __syncthreads();
    if (threadIdx.x < i)
      tmp[threadIdx.x] += tmp[i + threadIdx.x];
  }

  if (threadIdx.x == 0)
    partial[blockIdx.x] = tmp[0];
}

// Reduces the output of the vecdot_partial kernel to a single value. The result is stored in result.
__global__ void vecdot_reduce(double* partial, double* result)
{
  __shared__ double tmp[NUM_BLOCKS];

  if (threadIdx.x < NUM_BLOCKS)
    tmp[threadIdx.x] = partial[threadIdx.x];
  else
    tmp[threadIdx.x] = 0;

  for (int i=blockDim.x/2;i>=1;i = i/2) {
    __syncthreads();
    if (threadIdx.x < i)
      tmp[threadIdx.x] += tmp[i + threadIdx.x];
  }

  if (threadIdx.x == 0)
    *result = tmp[0];
}

// Divides num by den and stores the result in result. This is very wasteful of the GPU.
__global__ void scalardiv(double* num, double* den, double* result)
{
  if(threadIdx.x==0 && blockIdx.x==0)
    *result = (*num)/(*den);
}

// Computes r= a*x+y for n-length vectors x and y, and scalar a.
__global__ void axpy(int n, double* a, double* x, double* y, double* r)
{
  int elem;

  for (elem=THREAD_ID; elem<n; elem+=THREAD_COUNT)
    r[elem] = y[elem] + (*a)*x[elem];
}

// Computes y= y-a*x for n-length vectors x and y, and scalar a.
__global__ void ymax(int n, double* a, double* x, double* y)
{
  int elem;

  for (elem=THREAD_ID; elem<n; elem+=THREAD_COUNT)
    y[elem] = y[elem] - (*a)*x[elem];
}

// Convenient function for performing a vector dot product and reduce all in one go.
void vecdot(int n, double* vec1, double* vec2, double* result)
{
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);

  vecdot_partial<<<GridDim,BlockDim>>>(n, vec1, vec2, scratchpad);
  vecdot_reduce<<<1,NUM_BLOCKS>>>(scratchpad, result);
}

// Sets dest=src for scalars on the GPU.
void scalarassign(double* dest, double* src)
{
  hipMemcpy(dest, src, sizeof(double), hipMemcpyDeviceToDevice);
}

// Sets dest=src for n-length vectors on the GPU.
void vecassign(double *dest, double *src, int n)
{
  hipMemcpy(dest, src, sizeof(double)*n, hipMemcpyDeviceToDevice);
}


//void cg_solve(int* findrm, int size_findrm, int* colm, int size_colm, double* k_val, double* b, int rhs_size, double *x)

void cg_solve(int* k_findrm, int size_findrm, int* k_colm, int size_colm, double* k_val, double* k_b, int rhs_val_size, double *x_p)
{
  // Vectors on the GPU
  double
    //*k_x, *k_r,
    *k_d, *k_q, *k_s;
  // Diagonal matrix on the GPU (stored as a vector)
  double* k_jac;
  // Scalars on the GPU
  double  *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;

  // Scalars on the host
  double s0, snew;
  int iterations = 0;

  // Allocate space on the GPU for the CSR matrix and RHS vector, and copy from host to GPU
  hipBindTexture(NULL, tex_colm, k_colm, sizeof(int)*(size_colm));

  // Allocate space for vectors on the GPU
  //hipMalloc((void**)&k_x, sizeof(double)*(*rhs_val_size));
  //hipMalloc((void**)&k_r, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_s, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_d, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_q, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_jac, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_alpha, sizeof(double));
  hipMalloc((void**)&scratchpad, sizeof(double)*NUM_BLOCKS);
  hipMalloc((void**)&k_snew, sizeof(double));
  hipMalloc((void**)&k_sold, sizeof(double));
  hipMalloc((void**)&k_beta, sizeof(double));
  hipMalloc((void**)&k_s0, sizeof(double));

  // Dimensions of blocks and grid on the GPU
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);

  // Create diagonal preconditioning matrix (J = 1/diag(M)) 
  create_jac_sym<<<GridDim,BlockDim>>>(rhs_val_size, k_findrm, k_colm, k_val, k_jac);
  //  printd("jac", k_jac, 1000);
  // Bind the matrix to the texture cache - this was not done earlier as we modified the matrix
  hipBindTexture(NULL, tex_val, k_val, sizeof(double)*(size_colm));

  // Initialise result vector (x=0)
  veczero<<<GridDim,BlockDim>>>(rhs_val_size, x_p);

  // r=b-Ax (r=b since x=0), and d=M^(-1)r
  //hipMemcpy(k_r, k_b, sizeof(double)*(*rhs_val_size), hipMemcpyDeviceToDevice);
  //hipMemcpy(k_d, k_r, sizeof(double)*(*rhs_val_size), hipMemcpyDeviceToDevice);
  diag_spmv<<<GridDim,BlockDim>>>(rhs_val_size, k_jac, k_b, k_d);

  // s0 = r.d
  vecdot(rhs_val_size, k_b, k_d, k_s0);
  // snew = s0
  scalarassign(k_snew, k_s0);

  // Copy snew and s0 back to host so that host can evaluate stopping condition
  hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&s0, k_s0, sizeof(double), hipMemcpyDeviceToHost);
  // While i < imax and snew > epsilon^2*s0
  while (iterations < IMAX && snew > epsilon2*s0)
  {
    // q = Ad
    csr_spmv<<<GridDim,BlockDim>>>(rhs_val_size, k_d, k_q, k_findrm);
    // alpha = snew/(d.q)
    vecdot(rhs_val_size, k_d, k_q, k_alpha);
    scalardiv<<<1,1>>>(k_snew, k_alpha, k_alpha);
    // x = x + alpha*d
    axpy<<<GridDim,BlockDim>>>(rhs_val_size, k_alpha, k_d, x_p, x_p);
    // r = r - alpha*q
    ymax<<<GridDim,BlockDim>>>(rhs_val_size, k_alpha, k_q, k_b);
    // s = M^(-1)r
    diag_spmv<<<GridDim,BlockDim>>>(rhs_val_size, k_jac, k_b, k_s);
    // sold = snew
    scalarassign(k_sold, k_snew);
    // snew = r.s
    vecdot(rhs_val_size, k_b, k_s, k_snew);
    // beta = snew/sold
    scalardiv<<<1,1>>>(k_snew, k_sold, k_beta);
    // d = s + beta*d
    axpy<<<GridDim,BlockDim>>>(rhs_val_size, k_beta, k_d, k_s, k_d);
    // Copy back snew so the host can evaluate the stopping condition
    hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    // i = i+1
    iterations++;
  }

  hipUnbindTexture(tex_colm);
  hipUnbindTexture(tex_val);

  hipFree(k_s);
  hipFree(k_d);
  hipFree(k_q);
  hipFree(k_jac);
  hipFree(k_alpha);
  hipFree(k_snew);
  hipFree(k_sold);
  hipFree(k_beta);
  hipFree(k_s0);
  hipFree(scratchpad);

}

// LMA Addtions

__device__ int eleId(int ele, int node, int n) {
  return node*n+ele;
}

__device__ int lmatIdx(int a, int b, int c, int n) {
  return a*3*n+b*n+c;
}

// This definitely gives me the diagonal of the BD local matrix
__global__ void extract_diagonal(int n_ele, double *matrix, double *jac_tmp)
{
  for(int i=THREAD_ID; i<n_ele; i+=THREAD_COUNT) {
    jac_tmp[i*3  ] = matrix[lmatIdx(0,0,i,n_ele)];
    jac_tmp[i*3+1] = matrix[lmatIdx(1,1,i,n_ele)];
    jac_tmp[i*3+2] = matrix[lmatIdx(2,2,i,n_ele)];
  }
}

__global__ void create_jac_pc(int n, int *findrm, int *colm, double *jac, double *jac_tmp) {
  for(int row=THREAD_ID; row<n; row+=THREAD_COUNT) {
    jac[row] = 0;
    int a=findrm[row];
    int b=findrm[row+1];
    for(int k=a;k<b;k++) {
      jac[row] += jac_tmp[colm[k-1]-1];
    }
    jac[row] = 1.0/jac[row];
  }
}

__global__ void spmv_stage1_2(int n_ele, double *matrix, double *src, double *temp2, int *node_nums)
{

  //Compute M*s*b
  for(int ele=THREAD_ID; ele<n_ele; ele+=THREAD_COUNT) {

    double tmpa = src[node_nums[eleId(ele,0,n_ele)]-1];
    double tmpb = src[node_nums[eleId(ele,1,n_ele)]-1];
    double tmpc = src[node_nums[eleId(ele,2,n_ele)]-1];
    temp2[ele*3]   = matrix[lmatIdx(0,0,ele,n_ele)]*tmpa
                   + matrix[lmatIdx(0,1,ele,n_ele)]*tmpb
                   + matrix[lmatIdx(0,2,ele,n_ele)]*tmpc;
    temp2[ele*3+1] = matrix[lmatIdx(1,0,ele,n_ele)]*tmpa
                   + matrix[lmatIdx(1,1,ele,n_ele)]*tmpb
                   + matrix[lmatIdx(1,2,ele,n_ele)]*tmpc;
    temp2[ele*3+2] = matrix[lmatIdx(2,0,ele,n_ele)]*tmpa
                   + matrix[lmatIdx(2,1,ele,n_ele)]*tmpb
                   + matrix[lmatIdx(2,2,ele,n_ele)]*tmpc;
  }
}

__global__ void spmv_stage3(int nodes, double *temp2, double *dest, int *findrm, int *colm)
{
  // Compute s^T*M*s*b
  for(int row=THREAD_ID; row<nodes; row+=THREAD_COUNT) {
    dest[row] = 0;
    int a=findrm[row];
    int b=findrm[row+1];
    for(int k=a;k<b;k++)
      dest[row] += temp2[colm[k-1]-1];
  }
}

void cg_solve_lma(int* k_findrm, int size_findrm, int* k_colm, int size_colm, double* k_val, double* k_b, int rhs_val_size, double *x_p)
{
  // Vectors on the GPU
  double *k_d, *k_q, *k_s;
  double *temp1, *temp2, *k_jac_tmp;

  // Diagonal matrix on the GPU (stored as a vector)
  double* k_jac;

  // Scalars on the GPU
  double  *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;

  // Scalars on the host
  double s0, snew;
  int iterations = 0;

  // Allocate space for vectors on the GPU
  hipMalloc((void**)&k_s, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_d, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_q, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_jac, sizeof(double)*(rhs_val_size));
  hipMalloc((void**)&k_alpha, sizeof(double));
  hipMalloc((void**)&scratchpad, sizeof(double)*NUM_BLOCKS);
  hipMalloc((void**)&k_snew, sizeof(double));
  hipMalloc((void**)&k_sold, sizeof(double));
  hipMalloc((void**)&k_beta, sizeof(double));
  hipMalloc((void**)&k_s0, sizeof(double));
  hipMalloc((void**)&temp1, sizeof(double)*n_ele*3);
  hipMalloc((void**)&temp2, sizeof(double)*n_ele*3);
  hipMalloc((void**)&k_jac_tmp, sizeof(double)*n_ele*3);


  // Dimensions of blocks and grid on the GPU
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);

  // Create diagonal preconditioning matrix (J = 1/diag(M)) 
  extract_diagonal<<<GridDim,BlockDim>>>(n_ele, matrix, k_jac_tmp);
  create_jac_pc<<<GridDim,BlockDim>>>(rhs_val_size, k_findrm, k_colm, k_jac, k_jac_tmp);

  // Initialise result vector (x=0)
  veczero<<<GridDim,BlockDim>>>(rhs_val_size, x_p);

  // r=b-Ax (r=b since x=0), and d=M^(-1)r
  diag_spmv<<<GridDim,BlockDim>>>(rhs_val_size, k_jac, k_b, k_d);

  // s0 = r.d
  vecdot(rhs_val_size, k_b, k_d, k_s0);
  // snew = s0
  scalarassign(k_snew, k_s0);

  // Copy snew and s0 back to host so that host can evaluate stopping condition
  hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&s0, k_s0, sizeof(double), hipMemcpyDeviceToHost);
  
  // While i < imax and snew > epsilon^2*s0
  while (iterations < IMAX && snew > epsilon2*s0)
  {
    // q = Ad
    spmv_stage1_2<<<GridDim,BlockDim>>>(n_ele, matrix, k_d, temp2, node_nums);
    spmv_stage3<<<GridDim,BlockDim>>>(rhs_val_size, temp2, k_q, k_findrm, k_colm);
    // alpha = snew/(d.q)
    vecdot(rhs_val_size, k_d, k_q, k_alpha);
    scalardiv<<<1,1>>>(k_snew, k_alpha, k_alpha);
    // x = x + alpha*d
    axpy<<<GridDim,BlockDim>>>(rhs_val_size, k_alpha, k_d, x_p, x_p);
    // r = r - alpha*q
    ymax<<<GridDim,BlockDim>>>(rhs_val_size, k_alpha, k_q, k_b);
    // s = M^(-1)r
    diag_spmv<<<GridDim,BlockDim>>>(rhs_val_size, k_jac, k_b, k_s);
    // sold = snew
    scalarassign(k_sold, k_snew);
    // snew = r.s
    vecdot(rhs_val_size, k_b, k_s, k_snew);
    // beta = snew/sold
    scalardiv<<<1,1>>>(k_snew, k_sold, k_beta);
    // d = s + beta*d
    axpy<<<GridDim,BlockDim>>>(rhs_val_size, k_beta, k_d, k_s, k_d);
    // Copy back snew so the host can evaluate the stopping condition
    hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    // i = i+1
    iterations++;
  }

  hipFree(k_s);
  hipFree(k_d);
  hipFree(k_q);
  hipFree(k_jac);
  hipFree(k_alpha);
  hipFree(k_snew);
  hipFree(k_sold);
  hipFree(k_beta);
  hipFree(k_s0);
  hipFree(scratchpad);
  hipFree(temp1);
  hipFree(temp2);
  hipFree(k_jac_tmp);

}

